#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>

__global__ void nabeatsuKernel(bool* result, int result_len)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (i < result_len) {
        int val = i + 1;
        result[i] = (val % 3) == 0;

        while (val > 0) {
            result[i] |= (val % 10) == 3;
            val /= 10;
        }
    }
}

void showInfo() {
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);

    std::cout << "Devuce: " << devProp.name << std::endl;
    std::cout << "Global memory available on device in bytes: " << devProp.totalGlobalMem << std::endl;
    std::cout << "Shared memory available per block in bytes: " << devProp.sharedMemPerBlock << std::endl;
    std::cout << "Warp size in threads: " << devProp.warpSize << std::endl;
    std::cout << "Maximum number of threads per block: " << devProp.maxThreadsPerBlock << std::endl;
    std::cout << "Compute capacity: " << devProp.major << "." <<  devProp.minor << std::endl;
    std::cout << "Clock frequency in kilohertz: " << devProp.clockRate << std::endl;
    std::cout << "Number of multiprocessors on device: " << devProp.multiProcessorCount << std::endl;
}

int getMaxThreadsPerBlock() {
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);
    return devProp.maxThreadsPerBlock;
}

bool invokeNabeatsu(bool *result, int result_len, int nBlock, int nThread)
{
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipSetDevice failed!" << std::endl;
        return false;
    }

    bool* dev_result = NULL;
    cudaStatus = hipMalloc((void**)&dev_result, result_len * sizeof(bool));
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMalloc failed!" << std::endl;
        return false;
    }

    nabeatsuKernel <<<nBlock, nThread>>> (dev_result, result_len);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "nabeatsuKernel failed: " <<  hipGetErrorString(cudaStatus) << std::endl;
        hipFree(dev_result);
        return false;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipDeviceSynchronize failed!" << std::endl;
        hipFree(dev_result);
        return false;
    }

    cudaStatus = hipMemcpy(result, dev_result, result_len * sizeof(bool), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMemcpy failed!" << std::endl;
        hipFree(dev_result);
        return false;
    }

    hipFree(dev_result);
    hipDeviceReset();

    return true;
}
